#include "hip/hip_runtime.h"
﻿// CUDA-ядро для расчёта весового спектра кодовых слов
// Использует константную и shared-память

#include <hip/hip_runtime.h>
#include <cstdint>
#include <stdexcept> 
// В начале .cu-файла (после include)
#ifdef __cplusplus
extern "C" {
#endif
// 1. Константная память для упакованной порождающей матрицы
constexpr int MAX_CONST_ELEMENTS = 8192;  // максимум элементов (64KB / 8 байт)
__constant__ uint64_t d_matrix[MAX_CONST_ELEMENTS];

// 2. Глобальный массив спектра в глобальной памяти устройства
// d_spectrum выделяется и обнуляется на хосте

// 3. Ядро CUDA
__global__ void computeSpectrumKernel(uint64_t* d_spectrum,
    int n,
    int k,
    int blockCount,
    uint64_t numComb)
{
    // 3.1 Shared-память для локального спектра каждого блока
    extern __shared__ uint64_t s_spectrum[];  // размер = n+1

    int tid = threadIdx.x;
    int threadsPerBlock = blockDim.x;
    // инициализируем локальный спектр нулями
    for (int idx = tid; idx <= n; idx += threadsPerBlock) {
        s_spectrum[idx] = 0ULL;
    }
    __syncthreads();

    // 3.2 Вычисляем глобальный индекс и общее число потоков
    uint64_t globalTid = blockIdx.x * blockDim.x + tid;
    uint64_t totalThreads = gridDim.x * blockDim.x;

    // 3.3 Stride-цикл по всем комбинациям
    uint64_t codeword[16];  // предполагается blockCount<=16
    for (uint64_t mask = globalTid; mask < numComb; mask += totalThreads) {
        // обнуляем codeword
        for (int b = 0; b < blockCount; ++b) {
            codeword[b] = 0ULL;
        }
        // XOR строк из константной памяти
        for (int row = 0; row < k; ++row) {
            if (mask & (1ULL << row)) {
                int offset = row * blockCount;
                for (int b = 0; b < blockCount; ++b) {
                    codeword[b] ^= d_matrix[offset + b];
                }
            }
        }
        // подсчёт веса
        int weight = 0;
        for (int b = 0; b < blockCount; ++b) {
            weight += __popcll(codeword[b]);
        }
        // накапливаем в shared-памяти
        atomicAdd(&s_spectrum[weight], 1ULL);
    }
    __syncthreads();

    // 3.4 Сводим локальный спектр в глобальный
    if (tid == 0) {
        for (int i = 0; i <= n; ++i) {
            atomicAdd(&d_spectrum[i], s_spectrum[i]);
        }
    }
}

// 4. Хостовая обёртка для запуска
// Принимает: массив matrix на хосте, массив spectrum на хосте, параметры n,k, blockCount, threadsPerBlock, blocks
void launchSpectrumKernel(const uint64_t* h_matrixPacked,
    uint64_t* h_spectrum,
    int n,
    int k,
    int blockCount,
    int threadsPerBlock,
    int blocks)
{
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess || deviceCount == 0) {
        throw std::runtime_error("Error: CUDA-compatible GPU not found or CUDA is not available.");
    }
    // 4.1 Копируем порождающую матрицу в константную память устройства
    size_t matrixBytes = size_t(k) * blockCount * sizeof(uint64_t);
    hipMemcpyToSymbol(HIP_SYMBOL(d_matrix),
        h_matrixPacked,
        matrixBytes);

    // 4.2 Выделяем глобальный массив спектра на устройстве
    uint64_t* d_spectrum;
    size_t spectrumBytes = size_t(n + 1) * sizeof(uint64_t);
    hipMalloc(&d_spectrum, spectrumBytes);
    // Обнуляем спектр на устройстве
    hipMemset(d_spectrum, 0, spectrumBytes);

    // 4.3 Вычисляем число комбинаций
    uint64_t numComb = 1ULL << k;

    // 4.4 Размер shared-памяти для каждого блока
    size_t sharedBytes = spectrumBytes;

    // 4.5 Запуск ядра с заданными blocks и threadsPerBlock
    computeSpectrumKernel << <blocks, threadsPerBlock, sharedBytes >> > (
        d_spectrum,
        n,
        k,
        blockCount,
        numComb);

    // Ждём окончания выполнения на GPU
    hipDeviceSynchronize();

    // 4.6 Копируем результат обратно на хост
    hipMemcpy(h_spectrum, d_spectrum, spectrumBytes, hipMemcpyDeviceToHost);

    // 4.7 Освобождаем устройство
    hipFree(d_spectrum);
}
#ifdef __cplusplus
} // extern "C"
#endif